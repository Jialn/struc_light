
#include <hip/hip_runtime.h>

__global__ void cuda_test(float *dest, float *a, float *b, float *offset) // a simple test function
{
    const int idx = threadIdx.x +  blockIdx.x*blockDim.x;
    dest[idx] = a[idx] + b[idx] + offset[0];
}

// a function to convert RGGB bayer image to single blue channle image
__global__ void convert_bayer_to_blue(unsigned char *src, int *height_array, int *width_array)
{   
    int width = width_array[0], width_half = width_array[0] / 2, height_div2 = height_array[0] / 2;
    int idx_div4 = threadIdx.x + blockIdx.x*blockDim.x;
    int h_div2 = idx_div4 / width_half, w_div2 = idx_div4 % width_half;
    int idx = h_div2 * 2 * width + w_div2 * 2;
    if (h_div2 % height_div2 != 0) {    // if not the first line
        src[idx] = ((int)src[idx-width-1] + (int)src[idx-width+1] + (int)src[idx+width-1] + (int)src[idx+width+1]+2) / 4;  // R
        src[idx+1] = ((int)src[idx-width+1] + (int)src[idx+width+1]+1) /2;  //G
    }
    else {
        src[idx] = ((int)src[idx+width-1] + (int)src[idx+width+1]+1) / 2;   //R
        src[idx+1] = src[idx+width+1];                                      //G
    }
    src[idx+width] = ((int)src[idx+width-1] + (int)src[idx+width+1]+1) / 2; //G
    //src[idx+width+1] = src[idx+width+1]; //B
}

// a function to convert RGGB bayer image to single channle gray image
__global__ void convert_bayer_to_gray(unsigned char *src, int *height_array, int *width_array)
{   
    int width = width_array[0], width_half = width_array[0] / 2, height_div2 = height_array[0] / 2;
    int idx_div4 = threadIdx.x + blockIdx.x*blockDim.x;
    int h_div2 = idx_div4 / width_half, w_div2 = idx_div4 % width_half;
    int idx = h_div2 * 2 * width + w_div2 * 2;
    int r_value[4], b_value[4], g_value[4];
    int idx_r=idx, idx_g=idx+1, idx_g2=idx+width, idx_b=idx+width+1;
    if (h_div2 % height_div2 == 0) {    // if the first line
        // R
        r_value[0] = src[idx];
        b_value[0] = ((int)src[idx+width-1] + (int)src[idx+width+1]+1) / 2;
        if (w_div2 == 0) g_value[0] = ((int)src[idx+1] + (int)src[idx+width] + 1) / 2;
        else g_value[0] = ((int)src[idx-1] + (int)src[idx+1] + (int)src[idx+width] + 1) / 3;
        // G
        r_value[1] = ((int)src[idx_g-1] + (int)src[idx_g+1] + 1) / 2;
        g_value[1] = src[idx_g];
        b_value[1] = src[idx_g+width];
        // G
        r_value[2] = src[idx_g2+width];
        g_value[2] = src[idx_g2];
        b_value[2] = ((int)src[idx_g2-1] + (int)src[idx_g2+1] + 1) / 2;
        // B
        r_value[3] = ((int)src[idx_b+width-1] + (int)src[idx_b+width+1] + 1) / 2;
        g_value[3] = ((int)src[idx_b+width] + (int)src[idx_b-1] + (int)src[idx_b+1] + 2) / 3;
        b_value[3] = src[idx_b];
    }
    else if (h_div2 % height_div2 == height_div2-1) {   // if the last line
        // R
        r_value[0] = src[idx];
        g_value[0] = ((int)src[idx-width] + (int)src[idx-1] + (int)src[idx+1] + 1) / 3;
        if (w_div2 == width_half-1) b_value[0] = src[idx-width-1];
        else b_value[0] = ((int)src[idx-width-1] + (int)src[idx-width+1] + 1) / 2;
        // G
        r_value[1] = ((int)src[idx_g-1] + (int)src[idx_g+1] + 1) / 2;
        g_value[1] = src[idx_g];
        b_value[1] = ((int)src[idx_g-width] + (int)src[idx_g+width] + 1) /2;
        // G
        r_value[2] = src[idx_g2-width];
        g_value[2] = src[idx_g2];
        b_value[2] = ((int)src[idx_g2-1] + (int)src[idx_g2+1] + 1) /2;
        // B
        r_value[3] = ((int)src[idx_b-width-1] + (int)src[idx_b-width+1] + 1) / 2;
        g_value[3] = ((int)src[idx_b-width] + (int)src[idx_b-1] + (int)src[idx_b+1] + 2) / 4;
        b_value[3] = src[idx_b];

    }
    else {
        // R
        r_value[0] = src[idx];
        g_value[0] = ((int)src[idx-width] + (int)src[idx+width] + (int)src[idx-1] + (int)src[idx+1] + 2) / 4;
        b_value[0] = ((int)src[idx-width-1] + (int)src[idx-width+1] + (int)src[idx+width-1] + (int)src[idx+width+1] + 2) / 4;
        // G
        r_value[1] = ((int)src[idx_g-1] + (int)src[idx_g+1] + 1) / 2;
        g_value[1] = src[idx_g];
        b_value[1] = ((int)src[idx_g-width] + (int)src[idx_g+width] + 1) / 2;
        // G
        r_value[2] = ((int)src[idx_g2-width] + (int)src[idx_g2+width] + 1) / 2;
        g_value[2] = src[idx_g2];
        b_value[2] = ((int)src[idx_g2-1] + (int)src[idx_g2+1] + 1) / 2;
        // B
        r_value[3] = ((int)src[idx_b-width-1] + (int)src[idx_b-width+1] + (int)src[idx_b+width-1] + (int)src[idx_b+width+1] + 2) / 4;
        g_value[3] = ((int)src[idx_b-width] + (int)src[idx_b+width] + (int)src[idx_b-1] + (int)src[idx_b+1] + 2) / 4;
        b_value[3] = src[idx_b];
    }
    src[idx_r]  = (r_value[0] + g_value[0] + b_value[0] + 1) / 3;
    src[idx_g]  = (r_value[1] + g_value[1] + b_value[1] + 1) / 3;
    src[idx_g2] = (r_value[2] + g_value[2] + b_value[2] + 1) / 3;
    src[idx_b]  = (r_value[3] + g_value[3] + b_value[3] + 1) / 3;
}

__global__ void gray_decode(unsigned char *src, unsigned char *avg_thres_posi, unsigned char *avg_thres_nega, unsigned char *valid_map, int *image_num, int *height, int *width, short *img_index, int *unvalid_thres)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    bool pix_is_valid = ((int)avg_thres_posi[idx] - (int)avg_thres_nega[idx]) > unvalid_thres[0];
    // valid_map[idx] = pix_is_valid*255;  // if visulize valid map is needed
    if (! pix_is_valid) {
        img_index[idx] = -1;
        return;
    }
    int avg_thres = avg_thres_posi[idx]/2 + avg_thres_nega[idx]/2;
    int bin_code = 0;
    int current_bin_code_bit = 0;
    for (unsigned int i = 0; i < image_num[0]; i++) {
        int src_idx = idx + i * height[0] * width[0];
        if (src[src_idx]>=avg_thres) current_bin_code_bit = current_bin_code_bit ^ 1;
        else if (src[src_idx]<=avg_thres) current_bin_code_bit = current_bin_code_bit ^ 0;
        else {
            bin_code = -1;
            break;
        }
        bin_code += (current_bin_code_bit <<  (image_num[0]-1-i));
    }
    img_index[idx] = bin_code;
}

#define PI 3.14159265358979
__global__ void phase_shift_decode(unsigned char *src, int *height, int *width, float *img_phase, short *img_index, int *unvalid_thres, float *phsift_pattern_period_per_pixel_array)
{
    float phsift_pattern_period_per_pixel = phsift_pattern_period_per_pixel_array[0];
    float unvalid_thres_diff = unvalid_thres[0];
    float outliers_checking_diff_thres = 10.0 + unvalid_thres_diff; //above this, will skip outlier checking

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (img_index[idx] == -1) {
        img_phase[idx] = nanf("");
        img_index[idx] = 0;  //reuse img_index as belief map
        return;
    }
    float i1 = src[idx], i2 = src[idx + height[0] * width[0]];
    float i3 = src[idx + 2 * height[0] * width[0]], i4 = src[idx + 3 * height[0] * width[0]];
    bool unvalid_flag = (abs(i4 - i2) <= unvalid_thres_diff & abs(i3 - i1) <= unvalid_thres_diff);
    bool need_outliers_checking_flag = (abs(i4 - i2) <= outliers_checking_diff_thres & abs(i3 - i1) <= outliers_checking_diff_thres);
    if (unvalid_flag) {
        img_phase[idx] = nanf("");
        img_index[idx] = 0;
        return;
    }
    float phase = - atan2f(i4-i2, i3-i1) + PI;
    int phase_main_index = img_index[idx] / 2 ;
    int phase_sub_index = img_index[idx] & 0x01;
    if((phase_sub_index == 0) & (phase > PI*1.5))  phase -= 2.0*PI; 
    if((phase_sub_index == 1) & (phase < PI*0.5))  phase += 2.0*PI; 
    img_phase[idx] = phase_main_index * phsift_pattern_period_per_pixel + (phase * phsift_pattern_period_per_pixel / (2*PI));
    //reuse img_index as belief map
    if (need_outliers_checking_flag) img_index[idx] = 0;
    else img_index[idx] = abs(i4 - i2) + abs(i3 - i1);
}

__global__ void rectify_phase_and_belief_map(float *img_phase, short *bfmap, float *rectify_map_x, float *rectify_map_y, int *height_array, int *width_array, float *rectified_img_phase, short *rectified_bfmap, float *sub_pixel_map_x)
{
    const bool use_interpo_for_y_aixs = true;
    int width = width_array[0], height = height_array[0];
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int w = idx % width;
    float src_x = rectify_map_x[idx], src_y = rectify_map_y[idx];
    int round_y = int(src_y+0.499999), round_x = int(src_x+0.499999);
    int src_pix_idx = round_y*width + round_x;

    if (use_interpo_for_y_aixs) {
        int src_y_int = int(src_y);
        if (src_y_int == height-1) src_y_int = height - 2;
        float upper = img_phase[src_y_int*width+round_x];
        float lower = img_phase[src_y_int*width+round_x+width];
        float diff = lower - upper;
        if ( abs(diff) >= 1.0 | isnan(diff)) rectified_img_phase[idx] = img_phase[src_pix_idx];
        else rectified_img_phase[idx] = upper + diff * (src_y-src_y_int);
    }
    else rectified_img_phase[idx] = img_phase[src_pix_idx];
    rectified_bfmap[idx] = bfmap[src_pix_idx];
    sub_pixel_map_x[idx] = w + (round_x - src_x);
}

__device__ __forceinline__ void pix_index_matching(float *line_l, float *line_r, int w, int curr_pix_idx, int i, int line_start_addr_offset, float thres, short *belief_map_l, short *belief_map_r, int *most_corres_pts_l, int *most_corres_pts_r, int *most_corres_pts_l_bf, int *most_corres_pts_r_bf, int *cnt_l, int *cnt_r, float *average_corres_position_in_thres_l, float *average_corres_position_in_thres_r)
{
    if ((line_l[w]-thres <= line_r[i]) & (line_r[i] <= line_l[w])) {
        if (*most_corres_pts_l==-1) *most_corres_pts_l = i;
        else if (line_r[i] >= line_r[*most_corres_pts_l]) *most_corres_pts_l = i;
        #ifdef use_belief_map_for_checking
        int bfmap_thres = 10 + (belief_map_l[curr_pix_idx] + belief_map_r[line_start_addr_offset+i]) / 2;
        if (abs(belief_map_l[curr_pix_idx] - belief_map_r[line_start_addr_offset+i]) < bfmap_thres) {
            if (*most_corres_pts_l_bf==-1) *most_corres_pts_l_bf = i;
            else if (line_r[i] >= line_r[*most_corres_pts_l_bf]) *most_corres_pts_l_bf = i;
        }
        #endif
        *cnt_l += 1; 
        *average_corres_position_in_thres_l += i;
    }
    else if ((line_l[w] <= line_r[i]) & (line_r[i] <= line_l[w]+thres)) {
        if (*most_corres_pts_r==-1) *most_corres_pts_r = i;
        else if (line_r[i] <= line_r[*most_corres_pts_r]) *most_corres_pts_r = i;
        #ifdef use_belief_map_for_checking
        int bfmap_thres = 10 + (belief_map_l[curr_pix_idx] + belief_map_r[line_start_addr_offset+i]) / 2;
        if (abs(belief_map_l[curr_pix_idx] - belief_map_r[line_start_addr_offset+i]) < bfmap_thres) {
            if (*most_corres_pts_r_bf==-1) *most_corres_pts_r_bf = i;
            else if (line_r[i] <= line_r[*most_corres_pts_r_bf]) *most_corres_pts_r_bf = i;
        }
        #endif
        *cnt_r += 1;
        *average_corres_position_in_thres_r += i;
    }
}

__global__ void gen_depth_from_index_matching(float *depth_map, int *height_array, int *width_array, float *img_index_left, float *img_index_right, float *baseline,float *dmap_base,float *fx, float *img_index_left_sub_px,float *img_index_right_sub_px, short *belief_map_l, short *belief_map_r, float *roughly_projector_area_in_image, float *depth_cutoff, int *remove_possibly_outliers_when_matching)
{
    float depth_cutoff_near = depth_cutoff[0], depth_cutoff_far = depth_cutoff[1];
    int width = width_array[0];
    float projector_area_ratio = roughly_projector_area_in_image[0];
    float index_thres_for_matching = 1.5 * 1280.0 / (width*projector_area_ratio);  //the smaller projector_area in image, the larger index_offset cloud be
    int right_corres_point_offset_range = (1.333 * projector_area_ratio * width) / 128;
    bool check_outliers = (remove_possibly_outliers_when_matching[0] != 0);
    // if another pixel has similar index( < index_thres_for_outliers_checking) has a distance > max_allow_pixel_per_index, consider it's an outlier 
    float max_allow_pixel_per_index_for_outliers_checking = 2.5 + 1.0 * projector_area_ratio * width / 1280.0;
    float index_thres_for_outliers_checking = index_thres_for_matching * 1.2;

    int h = blockIdx.x, stride = blockDim.x, offset = threadIdx.x;  //blockIdx.x is current working line; blockDim.x is stride
    int thread_work_length = width / blockDim.y;  //blockDim.y is the num of threads group per line
    int start = thread_work_length*threadIdx.y, end = thread_work_length+start;
    int line_start_addr_offset = h * width;
    float *line_r = img_index_right + line_start_addr_offset, *line_l = img_index_left + line_start_addr_offset;
    int last_right_corres_point = -1;
    for (int w = start+offset; w < end; w+=stride) {
        int curr_pix_idx = line_start_addr_offset + w;
        depth_map[curr_pix_idx] = 0.0;
        if (isnan(line_l[w])) {
            last_right_corres_point = -1;
            continue;
        }
        // find the nearest left and right corresponding points in right image
        int most_corres_pts_l = -1, most_corres_pts_r = -1, most_corres_pts_l_bf = -1, most_corres_pts_r_bf = -1;
        int checking_left_edge = 0, checking_right_edge = width;
        int cnt_l = 0, cnt_r = 0;
        float average_corres_position_in_thres_l = 0, average_corres_position_in_thres_r = 0;
        if (last_right_corres_point > 0) {  // fast checking around last_right_corres_point
            checking_left_edge = last_right_corres_point - right_corres_point_offset_range;
            checking_right_edge = last_right_corres_point + right_corres_point_offset_range + stride;
            if (checking_left_edge <=0) checking_left_edge=0;
            if (checking_right_edge >=width) checking_right_edge=width;
            for (int i=checking_left_edge; i < checking_right_edge; i++) {
                if (isnan(line_r[i])) continue;
                float thres = index_thres_for_matching + abs(img_index_left_sub_px[line_start_addr_offset+w] - w - img_index_right_sub_px[line_start_addr_offset+i] + i)/projector_area_ratio;
                pix_index_matching(line_l, line_r, w, curr_pix_idx, i, line_start_addr_offset, thres, belief_map_l, belief_map_r, &most_corres_pts_l, &most_corres_pts_r, &most_corres_pts_l_bf, &most_corres_pts_r_bf, &cnt_l, &cnt_r, &average_corres_position_in_thres_l, &average_corres_position_in_thres_r);
            }
        }
        if (most_corres_pts_l == -1 & most_corres_pts_r == -1) {
            // last_right_corres_point is invalid or not found most_corres_pts, expand the searching range and try searching again
            for (int i=0; i < width; i++) { 
                if (isnan(line_r[i])) continue;
                float thres = index_thres_for_matching;
                pix_index_matching(line_l, line_r, w, curr_pix_idx, i, line_start_addr_offset, thres, belief_map_l, belief_map_r, &most_corres_pts_l, &most_corres_pts_r, &most_corres_pts_l_bf, &most_corres_pts_r_bf, &cnt_l, &cnt_r, &average_corres_position_in_thres_l, &average_corres_position_in_thres_r);
            }
        }
        // refine index of right 'w_r' by matching results
        float w_r = 0;
        bool outliers_flag = false;
        #ifdef use_belief_map_for_checking
        if (most_corres_pts_l_bf != -1) most_corres_pts_l = most_corres_pts_l_bf;
        if (most_corres_pts_r_bf != -1) most_corres_pts_r = most_corres_pts_r_bf;
        #endif
        if (most_corres_pts_l == -1 & most_corres_pts_r == -1) continue;
        else if (most_corres_pts_l==-1) w_r = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_r]+0.2; // add 0.2 pix offset as we know it's on the right side
        else if (most_corres_pts_r==-1) w_r = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_l]-0.2;
        else {
            // get the interpo right index 'w_r'
            float left_pos = line_r[most_corres_pts_l], right_pos = line_r[most_corres_pts_r];
            float left_value = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_l], right_value = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_r];
            if (right_pos-left_pos != 0) w_r = left_value + (right_value-left_value) * (line_l[w]-left_pos)/(right_pos-left_pos);
            else w_r = left_value;
        }
        if (cnt_l != 0) average_corres_position_in_thres_l = average_corres_position_in_thres_l / cnt_l;
        if (cnt_r != 0) average_corres_position_in_thres_r = average_corres_position_in_thres_r / cnt_r;
        // check possiblely outliers using max_allow_pixel_per_index and belief_map
        #ifdef use_belief_map_for_checking
        bool checkright = (most_corres_pts_r_bf==-1 | most_corres_pts_l_bf==-1);
        #else
        bool checkright = (belief_map_r[line_start_addr_offset+(int)(w_r+0.499999)]==0);
        #endif
        if (check_outliers & checkright) {  // & belief_map_r[line_start_addr_offset+(int)(w_r+0.499999)]==0
            if (most_corres_pts_l != -1 & abs((float)(most_corres_pts_l-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
            if (most_corres_pts_r != -1 & abs((float)(most_corres_pts_r-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
            if (average_corres_position_in_thres_l != 0 & abs((float)(average_corres_position_in_thres_l-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
            if (average_corres_position_in_thres_r != 0 & abs((float)(average_corres_position_in_thres_r-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
        }
        if (outliers_flag==true) continue;
        last_right_corres_point = (int)(w_r+0.499999);
        // get left index
        float w_l = img_index_left_sub_px[curr_pix_idx];
        // check possiblely left outliers
        #ifdef use_belief_map_for_checking
        bool checkleft = checkright; // | (belief_map_l[curr_pix_idx]==0);
        #else
        bool checkleft = (belief_map_l[curr_pix_idx]==0);
        #endif
        if (check_outliers & checkleft) {
            for (int i=0; i < width; i++) {
                if ((line_l[w]-index_thres_for_outliers_checking <= line_l[i]) & (line_l[i] <= line_l[w]+index_thres_for_outliers_checking)) {
                    if (abs((float)(w_l-i)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
                }
            }
        }
        if (outliers_flag==true) continue;
        // get stereo diff and depth
        float stereo_diff = dmap_base[0] + w_l - w_r;
        if (stereo_diff < 0) stereo_diff = - stereo_diff;
        if (stereo_diff > 0.000001) {
            float depth = fx[0] * baseline[0] / stereo_diff;
            if ((depth_cutoff_near < depth) & (depth < depth_cutoff_far)) depth_map[curr_pix_idx] = depth;
        }
    }
}

__global__ void optimize_dmap_using_sub_pixel_map(float *depth_map, float *optimized_depth_map, int *height_array, int *width_array, float *img_index_left_sub_px)
{
    // interpo for depth map using sub-pixel map
    // this does not improve a lot on rendered data because no distortion and less stereo rectify for left camera, but very useful for real captures
    int width = width_array[0];
    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int w = current_pix_idx % width;
    if (w == 0 | w == width-1) {
        optimized_depth_map[current_pix_idx] = depth_map[current_pix_idx];
        return;
    }

    float left_value = 0.0, right_value = 0.0;
    float left_pos = 0.0, right_pos = 0.0;
    float real_pos_for_current_depth = img_index_left_sub_px[current_pix_idx];
    if (depth_map[current_pix_idx] <= 0.00001) {
        if (depth_map[current_pix_idx-1] >= 0.00001 & depth_map[current_pix_idx+1] >= 0.00001) {
            right_pos = img_index_left_sub_px[current_pix_idx+1];
            right_value = depth_map[current_pix_idx+1];
            left_pos = img_index_left_sub_px[current_pix_idx-1];
            left_value = depth_map[current_pix_idx-1];
        }
    }
    else if (real_pos_for_current_depth >= w) {
        right_pos = real_pos_for_current_depth;
        right_value = depth_map[current_pix_idx];
        left_pos = img_index_left_sub_px[current_pix_idx-1];
        left_value = depth_map[current_pix_idx-1];
    }
    else {
        right_pos = img_index_left_sub_px[current_pix_idx+1];
        right_value = depth_map[current_pix_idx+1];
        left_pos = real_pos_for_current_depth;
        left_value = depth_map[current_pix_idx];
    }
    if (left_value >= 0.00001 & right_value >= 0.00001) {
        optimized_depth_map[current_pix_idx] = left_value + (right_value-left_value) * (w-left_pos)/(right_pos-left_pos);
    }
    else {
        optimized_depth_map[current_pix_idx] = 0.0;
    }
}

// flying points filter
// a point could be considered as not flying when: points in checking range below max_distance > minmum num
__global__ void flying_points_filter(float *depth_map, float *depth_map_raw, int *height_array, int *width_array, float *camera_kp, float *depth_filter_max_distance, int *depth_filter_minmum_points_in_checking_range, int *belief_map)
{
    #define use_fast_distance_checking_for_flying_points_filter // use 3D distance (slower but more precisely) or only distance of axis-z to check flying points
            // enable this will save above 95% time compared with 3D distance checking, while can still remove most of the flying pts.
            // an example (3d vs only_z): render0000_2k avg error @ 10 mm thres: 0.1145mm vs 0.1152mm; cost time: 17ms vs 1ms on TitanRTX
    int height = height_array[0];
    int width = width_array[0];
    float max_distance = depth_filter_max_distance[0];
    int minmum_point_num_in_range = depth_filter_minmum_points_in_checking_range[0] + (width / 400) * (width / 400);
    float checking_range_in_meter = max_distance * 1.2;
    int checking_range_limit = width/50;
    float fx = camera_kp[0];
    float cx = camera_kp[2];
    float fy = camera_kp[1*3+1];
    float cy = camera_kp[1*3+2];

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map_raw[current_pix_idx];
    depth_map[current_pix_idx] = curr_pix_value;

    if (curr_pix_value != 0) {
        float point_x = curr_pix_value * (w - cx) / fx;
        float point_y = curr_pix_value * (h - cy) / fy;
        int checking_range_in_pix_x = (int)(checking_range_in_meter * fx / curr_pix_value);
        int checking_range_in_pix_y = (int)(checking_range_in_meter * fy / curr_pix_value);
        checking_range_in_pix_x = min(checking_range_in_pix_x, checking_range_limit);
        checking_range_in_pix_y = min(checking_range_in_pix_y, checking_range_limit);
        int is_not_flying_point_flag = 0;
        
        for (unsigned int i = max(0, h-checking_range_in_pix_y); i < min(height, h+checking_range_in_pix_y+1); i++) {
            int line_i_offset = i * width;
            for (unsigned int j = max(0, w-checking_range_in_pix_x); j < min(width, w+checking_range_in_pix_x+1); j++) {
                float checking_pix_value = depth_map_raw[line_i_offset + j];
                float z_diff = abs(curr_pix_value - checking_pix_value);
                if (checking_pix_value != 0.0 & z_diff < max_distance) {
                    #ifndef use_fast_distance_checking_for_flying_points_filter
                    float curr_x = checking_pix_value * (j - cx) / fx;
                    float curr_y = checking_pix_value * (i - cy) / fy;
                    float x_diff = curr_x - point_x, y_diff = curr_y - point_y;
                    float distance = (x_diff)*(x_diff) + (y_diff)*(y_diff) + (z_diff)*(z_diff);
                    if (distance < max_distance*max_distance) is_not_flying_point_flag += 1;
                    #else
                    is_not_flying_point_flag += 1;
                    #endif
                }
            }
            if (is_not_flying_point_flag > minmum_point_num_in_range) break;
        }
        if (is_not_flying_point_flag <= minmum_point_num_in_range) depth_map[current_pix_idx] = 0.0;
    }
}

__global__ void depth_filter_w(float *depth_map_out, float *depth_map, int *height_array, int *width_array, int *depth_filter_max_length, float *depth_filter_unvalid_thres, short *belief_map)
{
    int height = height_array[0];
    int width = width_array[0];
    int filter_max_length = depth_filter_max_length[0];
    float filter_thres = depth_filter_unvalid_thres[0];
    const float filter_weights[5] = {1.0, 0.6, 0.4, 0.2, 0.1};

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map[current_pix_idx];
    if (curr_pix_value != 0) {
        int line_start_addr_offset = h * width;
        // horizontal
        float left_weight = 0.0, right_weight = 0.0, depth_sum = curr_pix_value*filter_weights[0];
        for (int i=1; i< filter_max_length+1; i++) {
            int l_idx = w-i, r_idx = w+i;
            if (!(l_idx > 0 & r_idx < width)) break;
            // if (belief_map[current_pix_idx] >= 1) filter_thres = depth_filter_unvalid_thres[0];
            // else filter_thres = depth_filter_unvalid_thres[0] * 4;
            else if (depth_map[line_start_addr_offset+l_idx] != 0 & depth_map[line_start_addr_offset+r_idx] != 0 & \
            abs(depth_map[line_start_addr_offset+l_idx] - curr_pix_value) < filter_thres & abs(depth_map[line_start_addr_offset+r_idx] - curr_pix_value) < filter_thres) {

                left_weight += filter_weights[i];
                right_weight += filter_weights[i];
                depth_sum += (depth_map[line_start_addr_offset+r_idx] + depth_map[line_start_addr_offset+l_idx]) * filter_weights[i];
            }
            else continue;
        }
        depth_map_out[current_pix_idx] = depth_sum / (filter_weights[0] + left_weight + right_weight);
    }
    else {
        depth_map_out[current_pix_idx] = depth_map[current_pix_idx];
    }
}

__global__ void depth_filter_h(float *depth_map_out, float *depth_map, int *height_array, int *width_array, int *depth_filter_max_length, float *depth_filter_unvalid_thres, short *belief_map)
{
    int height = height_array[0];
    int width = width_array[0];
    int filter_max_length = depth_filter_max_length[0];
    float filter_thres = depth_filter_unvalid_thres[0];
    const float filter_weights[5] = {1.0, 0.6, 0.4, 0.2, 0.1};

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map[current_pix_idx];
    if (curr_pix_value != 0) {
        int line_start_addr_offset = h * width;
        // horizontal
        float left_weight = 0.0, right_weight = 0.0, depth_sum = curr_pix_value*filter_weights[0];
        for (int i=1; i< filter_max_length+1; i++) {
            int l_idx = h-i, r_idx = h+i;
            if (!(l_idx > 0 & r_idx < width)) break;
            else if (depth_map[l_idx*width+w] != 0 & depth_map[r_idx*width+w] != 0 & \
                    abs(depth_map[l_idx*width+w] - curr_pix_value) < filter_thres & abs(depth_map[r_idx*width+w] - curr_pix_value) < filter_thres) {
                left_weight += filter_weights[i];
                right_weight += filter_weights[i];
                depth_sum += (depth_map[r_idx*width+w] + depth_map[l_idx*width+w]) * filter_weights[i];
            }
            else continue;
        }
        depth_map_out[current_pix_idx] = depth_sum / (filter_weights[0] + left_weight + right_weight);
    }
    else {
        depth_map_out[current_pix_idx] = depth_map[current_pix_idx];
    }
}

__device__ __forceinline__ float get_mid(float a, float b, float c)
{
    float max=a, min=a;
    if (b > max) max = b;
    if (c > max) max = c;
    if (b < min) min = b;
    if (c < min) min = c;
    return a+b+c-min-max;
}

__global__ void depth_median_filter_w(float *depth_map_out, float *depth_map, int *height_array, int *width_array, int *depth_filter_max_length)
{   // filter_max_length (ksize) fixed to 1 for now
    int height = height_array[0];
    int width = width_array[0];
    int filter_max_length = 1; //depth_filter_max_length[0]; // 1, 2

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map[current_pix_idx];
    float mid_val = curr_pix_value;
    if (curr_pix_value != 0 & h != 0 & h!= height-1 & w !=0 & w != width-1) {
        if(depth_map[current_pix_idx-1] != 0 & depth_map[current_pix_idx+1] != 0) {
            mid_val = get_mid(depth_map[current_pix_idx-1], depth_map[current_pix_idx], depth_map[current_pix_idx+1]);
        }
    }
    depth_map_out[current_pix_idx] = mid_val;
}

__global__ void depth_median_filter_h(float *depth_map_out, float *depth_map, int *height_array, int *width_array, int *depth_filter_max_length)
{
    int height = height_array[0];
    int width = width_array[0];
    int filter_max_length = 1;

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map[current_pix_idx];
    float mid_val = curr_pix_value;
    if (curr_pix_value != 0 & h != 0 & h!= height-1 & w !=0 & w != width-1) {
        if(depth_map[current_pix_idx-width] != 0 & depth_map[current_pix_idx+width] != 0) {
            mid_val = get_mid(depth_map[current_pix_idx-width], depth_map[current_pix_idx], depth_map[current_pix_idx+width]);
        }
    }
    depth_map_out[current_pix_idx] = mid_val;
}

__global__ void convert_dmap_to_mili_meter(float *depth_map)
{
    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    depth_map[current_pix_idx] = 1000.0*depth_map[current_pix_idx];
}
