
#include <hip/hip_runtime.h>

__global__ void multiply(float *dest, float *a, float *b, float *offset) // a test function
{
    const int idx = threadIdx.x +  blockIdx.x*blockDim.x;
    dest[idx] = a[idx] * b[idx] + offset[0];
}

__global__ void gray_decode(unsigned char *src, unsigned char *imgs_thresh, unsigned char *valid_map, int *image_num, int *height, int *width, short *img_index, int *unvalid_thres)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (valid_map[idx] == 0) {
        img_index[idx] = -1;
        return;
    }
    int bin_code = 0;
    int current_bin_code_bit = 0;
    for (unsigned int i = 0; i < image_num[0]; i++) {
        int src_idx = idx + i * height[0] * width[0];
        if (src[src_idx]>=imgs_thresh[idx]+unvalid_thres[0]) current_bin_code_bit = current_bin_code_bit ^ 1;
        else if (src[src_idx]<=imgs_thresh[idx]-unvalid_thres[0]) current_bin_code_bit = current_bin_code_bit ^ 0;
        else {
            bin_code = -1;
            break;
        }
        bin_code += (current_bin_code_bit <<  (image_num[0]-1-i));
    img_index[idx] = bin_code;
    }
}

__global__ void phase_shift_decode(unsigned char *src, int *height, int *width, float *img_phase, short *img_index, int *unvalid_thres)
{
    float phsift_pattern_period_per_pixel = 10.0;
    float unvalid_thres_diff = unvalid_thres[0];
    float outliers_checking_thres_diff = 4 * (1.0+unvalid_thres_diff); //above this, will skip outlier checking
    float pi = 3.14159265358979;

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (img_index[idx] == -1) {
        img_phase[idx] = nanf("");
        return;
    }
    float i1 = src[idx];
    float i2 = src[idx + height[0] * width[0]];
    float i3 = src[idx + 2 * height[0] * width[0]];
    float i4 = src[idx + 3 * height[0] * width[0]];
    bool unvalid_flag = (abs(i4 - i2) <= unvalid_thres_diff & abs(i3 - i1) <= unvalid_thres_diff);
    bool need_outliers_checking_flag = (abs(i4 - i2) <= outliers_checking_thres_diff & abs(i3 - i1) <= outliers_checking_thres_diff);
    if (unvalid_flag) {
        img_phase[idx] = nanf("");
        return;
    }
    float phase = - atan2f(i4-i2, i3-i1) + pi;
    int phase_main_index = img_index[idx] / 2 ;
    int phase_sub_index = img_index[idx] & 0x01;
    if((phase_sub_index == 0) && (phase > pi*1.5))  phase -= 2.0*pi; 
    if((phase_sub_index == 1) && (phase < pi*0.5))  phase += 2.0*pi; 
    img_phase[idx] = phase_main_index * phsift_pattern_period_per_pixel + (phase * phsift_pattern_period_per_pixel / (2*pi));
    img_index[idx] = not need_outliers_checking_flag;  //reuse img_index as belief map
}

__global__ void depth_filter(float *depth_map, float *depth_map_raw, int *height_array, int *width_array, float *camera_kp, float *depth_filter_max_distance, int *depth_filter_minmum_points_in_checking_range)
{
    // a point could be considered as not flying when: points in checking range below max_distance > minmum num 
    int height = height_array[0];
    int width = width_array[0];
    float max_distance = depth_filter_max_distance[0];
    int minmum_point_num_in_range = depth_filter_minmum_points_in_checking_range[0] + (width / 400) * (width / 400);
    float checking_range_in_meter = max_distance * 1.2;
    int checking_range_limit = width/50;
    float fx = camera_kp[0];
    float cx = camera_kp[2];
    float fy = camera_kp[1*3+1];
    float cy = camera_kp[1*3+2];

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;

    if (depth_map_raw[current_pix_idx] != 0) {
        float point_x = depth_map_raw[current_pix_idx] * (w - cx) / fx;
        float point_y = depth_map_raw[current_pix_idx] * (h - cy) / fy;
        int checking_range_in_pix_x = (int)(checking_range_in_meter * fx / depth_map_raw[current_pix_idx]);
        int checking_range_in_pix_y = (int)(checking_range_in_meter * fy / depth_map_raw[current_pix_idx]);
        checking_range_in_pix_x = min(checking_range_in_pix_x, checking_range_limit);
        checking_range_in_pix_y = min(checking_range_in_pix_y, checking_range_limit);
        int is_not_flying_point_flag = 0;
        
        for (unsigned int i = max(0, h-checking_range_in_pix_y); i < min(height, h+checking_range_in_pix_y+1); i++) {
            for (unsigned int j = max(0, w-checking_range_in_pix_x); j < min(width, w+checking_range_in_pix_x+1); j++) {
                int pix_ij_idx = i*width + j;
                float curr_x = depth_map_raw[pix_ij_idx] * (j - cx) / fx;
                float curr_y = depth_map_raw[pix_ij_idx] * (i - cy) / fy;
                float distance = (curr_x - point_x)*(curr_x - point_x) + (curr_y - point_y)*(curr_y - point_y) + (depth_map_raw[current_pix_idx] - depth_map_raw[pix_ij_idx])*(depth_map_raw[current_pix_idx] - depth_map_raw[pix_ij_idx]);
                if (distance < max_distance*max_distance) is_not_flying_point_flag += 1;
                if (is_not_flying_point_flag > minmum_point_num_in_range) break;
            }
        }
        
        if (is_not_flying_point_flag <= minmum_point_num_in_range) depth_map[current_pix_idx] = 0.0;
    }
}

__global__ void get_dmap_from_index_map(float *depth_map, int *height_array, int *width_array, float *img_index_left, float *img_index_right, float *baseline,float *dmap_base,float *fx, float *img_index_left_sub_px,float *img_index_right_sub_px, short *belief_map_l, short *belief_map_r, float *roughly_projector_area_in_image)
{
    float depth_cutoff_near = 0.1, depth_cutoff_far = 2.0;
    int width = width_array[0];
    float area_scale = 1.333 * roughly_projector_area_in_image[0];
    float max_allow_pixel_per_index = 1.25 + area_scale * width / 1280.0;
    float max_index_offset_when_matching = 1.3 * (1280.0 / width);  //typical condition: a lttle larger than 2.0 for 640, 1.0 for 1280, 0.5 for 2560
    float max_index_offset_when_matching_ex = max_index_offset_when_matching * 1.5;
    float right_corres_point_offset_range = (width / 128) * area_scale;
    bool check_outliers = true;

    int h = threadIdx.x + blockIdx.x*blockDim.x;  //current_line
    float *line_r = img_index_right + h * width;
    float *line_l = img_index_left + h * width;
    int last_right_corres_point = -1;
    for (int w = 0; w < width; w++) {
        if (isnan(line_l[w])) {
            last_right_corres_point = -1;
            continue;
        }
        int curr_pix_idx = h*width + w;
        // find the nearest left and right corresponding points in right image
        int cnt_l = 0, cnt_r = 0;
        int most_corres_pts_l = -1, most_corres_pts_r = -1;
        int checking_left_edge = 0, checking_right_edge = width;
        if (last_right_corres_point > 0) {
            checking_left_edge = last_right_corres_point - right_corres_point_offset_range;
            checking_right_edge = last_right_corres_point + right_corres_point_offset_range;
            if (checking_left_edge <=0) checking_left_edge=0;
            if (checking_right_edge >=width) checking_right_edge=width;
        }
        for (int i=checking_left_edge; i < checking_right_edge; i++) {
            if (isnan(line_r[i])) continue;
            if ((line_l[w]-max_index_offset_when_matching <= line_r[i]) & (line_r[i] <= line_l[w])) {
                if (most_corres_pts_l==-1) most_corres_pts_l = i;
                else if (line_l[w] - line_r[i] <= line_l[w] - line_r[most_corres_pts_l]) most_corres_pts_l = i;
                cnt_l += 1;
            }
            if ((line_l[w] <= line_r[i]) & (line_r[i] <= line_l[w]+max_index_offset_when_matching)) {
                if (most_corres_pts_r==-1) most_corres_pts_r = i;
                else if (line_r[i] - line_l[w] <= line_r[most_corres_pts_r] - line_l[w]) most_corres_pts_r = i;
                cnt_r += 1;
            }
        }
        if (cnt_l == 0 & cnt_r == 0) { // expand the searching range and try again
            for (int i=0; i < width; i++) { 
                if (isnan(line_r[i])) continue;
                if ((line_l[w]-max_index_offset_when_matching_ex <= line_r[i]) & (line_r[i] <= line_l[w])) {
                    if (most_corres_pts_l==-1) most_corres_pts_l = i;
                    else if (line_l[w] - line_r[i] <= line_l[w] - line_r[most_corres_pts_l]) most_corres_pts_l = i;
                    cnt_l += 1;
                }
                if ((line_l[w] <= line_r[i]) & (line_r[i] <= line_l[w]+max_index_offset_when_matching_ex)) {
                    if (most_corres_pts_r==-1) most_corres_pts_r = i;
                    else if (line_r[i] - line_l[w] <= line_r[most_corres_pts_r] - line_l[w]) most_corres_pts_r = i;
                    cnt_r += 1;
                }
            }
        }
        if (cnt_l == 0 & cnt_r == 0) continue;
        // get the interpo right index 'w_r'
        float w_r = 0;
        float left_pos = line_r[most_corres_pts_l], right_pos = line_r[most_corres_pts_r];
        float left_value = img_index_right_sub_px[h*width+most_corres_pts_l], right_value = img_index_right_sub_px[h*width+most_corres_pts_r];
        if ((cnt_l != 0) & (cnt_r != 0)) {
            if (right_pos-left_pos != 0) w_r = left_value + (right_value-left_value) * (line_l[w]-left_pos)/(right_pos-left_pos);
            else w_r = left_value;
        }
        else if (cnt_l != 0)    w_r = left_value;
        else                    w_r = right_value;
        // check possiblely outliers using max_allow_pixel_per_index and belief_map
        bool outliers_flag = false;
        if (check_outliers==true & belief_map_r[h*width+(int)round(w_r)]==0) {
            if (abs((float)(most_corres_pts_l-w_r)) > max_allow_pixel_per_index) outliers_flag = true;
            if (abs((float)(most_corres_pts_r-w_r)) > max_allow_pixel_per_index) outliers_flag = true;
        }
        if (outliers_flag==true) continue;
        last_right_corres_point = round(w_r);
        // get left index
        float w_l = img_index_left_sub_px[curr_pix_idx];
        // check possiblely left outliers
        if (check_outliers==true & belief_map_l[curr_pix_idx]==0) {
            for (int i=0; i < width; i++) {
                if ((line_l[w]-max_index_offset_when_matching_ex <= line_l[i]) & (line_l[i] <= line_l[w]+max_index_offset_when_matching_ex)) {
                    if (abs((float)(w-i)) > max_allow_pixel_per_index) outliers_flag = true;
                }
            }
        }
        if (outliers_flag==true) continue;
        // get stereo diff and depth
        float stereo_diff = dmap_base[0] + w_l - w_r;
        if (stereo_diff > 0.000001) {
            float depth = fx[0] * baseline[0] / stereo_diff;
            if ((depth_cutoff_near < depth) & (depth < depth_cutoff_far)) depth_map[curr_pix_idx] = depth;
        }
    }
}
