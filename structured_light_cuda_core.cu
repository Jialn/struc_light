
#include <hip/hip_runtime.h>

__global__ void cuda_test(float *dest, float *a, float *b, float *offset) // a simple test function
{
    const int idx = threadIdx.x +  blockIdx.x*blockDim.x;
    dest[idx] = a[idx] + b[idx] + offset[0];
}

__global__ void gray_decode(unsigned char *src, unsigned char *avg_thres_posi, unsigned char *avg_thres_nega, unsigned char *valid_map, int *image_num, int *height, int *width, short *img_index, int *unvalid_thres)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (valid_map[idx] == 0) {
        img_index[idx] = -1;
        return;
    }
    int avg_thres = avg_thres_posi[idx]/2 + avg_thres_nega[idx]/2;
    int bin_code = 0;
    int current_bin_code_bit = 0;
    for (unsigned int i = 0; i < image_num[0]; i++) {
        int src_idx = idx + i * height[0] * width[0];
        if (src[src_idx]>=avg_thres+unvalid_thres[0]) current_bin_code_bit = current_bin_code_bit ^ 1;
        else if (src[src_idx]<=avg_thres-unvalid_thres[0]) current_bin_code_bit = current_bin_code_bit ^ 0;
        else {
            bin_code = -1;
            break;
        }
        bin_code += (current_bin_code_bit <<  (image_num[0]-1-i));
    }
    img_index[idx] = bin_code;
}

__global__ void phase_shift_decode(unsigned char *src, int *height, int *width, float *img_phase, short *img_index, int *unvalid_thres, float *phsift_pattern_period_per_pixel_array)
{
    float phsift_pattern_period_per_pixel = phsift_pattern_period_per_pixel_array[0];
    float unvalid_thres_diff = unvalid_thres[0];
    float outliers_checking_diff_thres = 10.0 + unvalid_thres_diff; //above this, will skip outlier checking
    const float pi = 3.14159265358979;

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (img_index[idx] == -1) {
        img_phase[idx] = nanf("");
        img_index[idx] = 0;  //reuse img_index as belief map
        return;
    }
    float i1 = src[idx];
    float i2 = src[idx + height[0] * width[0]];
    float i3 = src[idx + 2 * height[0] * width[0]];
    float i4 = src[idx + 3 * height[0] * width[0]];
    bool unvalid_flag = (abs(i4 - i2) <= unvalid_thres_diff & abs(i3 - i1) <= unvalid_thres_diff);
    bool need_outliers_checking_flag = (abs(i4 - i2) <= outliers_checking_diff_thres & abs(i3 - i1) <= outliers_checking_diff_thres);
    if (unvalid_flag) {
        img_phase[idx] = nanf("");
        img_index[idx] = 0;  //reuse img_index as belief map
        return;
    }
    float phase = - atan2f(i4-i2, i3-i1) + pi;
    int phase_main_index = img_index[idx] / 2 ;
    int phase_sub_index = img_index[idx] & 0x01;
    if((phase_sub_index == 0) & (phase > pi*1.5))  phase -= 2.0*pi; 
    if((phase_sub_index == 1) & (phase < pi*0.5))  phase += 2.0*pi; 
    img_phase[idx] = phase_main_index * phsift_pattern_period_per_pixel + (phase * phsift_pattern_period_per_pixel / (2*pi));
    //reuse img_index as belief map
    if (need_outliers_checking_flag) img_index[idx] = 0;
    else img_index[idx] = abs(i4 - i2) + abs(i3 - i1);  //reuse img_index as belief map, last bit is need_outliers_checking_flag
}

__global__ void rectify_phase_and_belief_map(float *img_phase, short *bfmap, float *rectify_map_x, float *rectify_map_y, int *height_array, int *width_array, float *rectified_img_phase, short *rectified_bfmap, float *sub_pixel_map_x)
{
    const bool use_interpo_for_y_aixs = true;
    int width = width_array[0], height = height_array[0];
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int w = idx % width;
    float src_x = rectify_map_x[idx], src_y = rectify_map_y[idx];
    int round_y = int(src_y+0.499999), round_x = int(src_x+0.499999);
    int src_pix_idx = round_y*width + round_x;

    if (use_interpo_for_y_aixs) {
        int src_y_int = int(src_y);
        if (src_y_int == height-1) src_y_int = height - 2;
        float upper = img_phase[src_y_int*width+round_x];
        float lower = img_phase[src_y_int*width+round_x+width];
        float diff = lower - upper;
        if ( abs(diff) >= 1.0 | isnan(diff)) rectified_img_phase[idx] = img_phase[src_pix_idx];
        else rectified_img_phase[idx] = upper + diff * (src_y-src_y_int);
    }
    else rectified_img_phase[idx] = img_phase[src_pix_idx];
    rectified_bfmap[idx] = bfmap[src_pix_idx];
    sub_pixel_map_x[idx] = w + (round_x - src_x);
}

__global__ void gen_depth_from_index_matching(float *depth_map, int *height_array, int *width_array, float *img_index_left, float *img_index_right, float *baseline,float *dmap_base,float *fx, float *img_index_left_sub_px,float *img_index_right_sub_px, short *belief_map_l, short *belief_map_r, float *roughly_projector_area_in_image, float *depth_cutoff, int *remove_possibly_outliers_when_matching)
{
    float depth_cutoff_near = depth_cutoff[0], depth_cutoff_far = depth_cutoff[1];
    int width = width_array[0];
    float projector_area_ratio = roughly_projector_area_in_image[0];
    float index_thres_for_matching = 1.5 * 1280.0 / (width*projector_area_ratio);  //the smaller projector_area in image, the larger index_offset cloud be
    int right_corres_point_offset_range = (1.333 * projector_area_ratio * width) / 128;
    bool check_outliers = (remove_possibly_outliers_when_matching[0] != 0);
    // if another pixel has similar index( < index_thres_for_outliers_checking) has a distance > max_allow_pixel_per_index, consider it's an outlier 
    float max_allow_pixel_per_index_for_outliers_checking = 2.5 + 1.0 * projector_area_ratio * width / 1280.0;
    float index_thres_for_outliers_checking = index_thres_for_matching * 1.2;
    const bool use_belief_map_checking_when_matching = false;

    int h = blockIdx.x, stride = blockDim.x, offset = threadIdx.x;  //blockIdx.x is current working line; blockDim.x is stride
    int thread_work_length = width / blockDim.y;  //blockDim.y is the num of threads group per line
    int start = thread_work_length*threadIdx.y, end = thread_work_length+start;
    int line_start_addr_offset = h * width;
    float *line_r = img_index_right + line_start_addr_offset, *line_l = img_index_left + line_start_addr_offset;
    int last_right_corres_point = -1;
    for (int w = start+offset; w < end; w+=stride) {
        int curr_pix_idx = line_start_addr_offset + w;
        depth_map[curr_pix_idx] = 0.0;
        if (isnan(line_l[w])) {
            last_right_corres_point = -1;
            continue;
        }
        // find the nearest left and right corresponding points in right image
        int most_corres_pts_l = -1, most_corres_pts_r = -1;
        int checking_left_edge = 0, checking_right_edge = width;
        int cnt_l = 0, cnt_r = 0;
        float average_corres_position_in_thres_l = 0, average_corres_position_in_thres_r = 0;
        if (last_right_corres_point > 0) {
            checking_left_edge = last_right_corres_point - right_corres_point_offset_range;
            checking_right_edge = last_right_corres_point + right_corres_point_offset_range + stride;
            if (checking_left_edge <=0) checking_left_edge=0;
            if (checking_right_edge >=width) checking_right_edge=width;
            for (int i=checking_left_edge; i < checking_right_edge; i++) {  // fast checking around last_right_corres_point
                if (isnan(line_r[i])) continue;
                if (use_belief_map_checking_when_matching) {
                    float bfmap_thres = 10 + (belief_map_l[curr_pix_idx] + belief_map_r[line_start_addr_offset+i]) * 0.75;
                    if (abs(belief_map_l[curr_pix_idx] - belief_map_r[line_start_addr_offset+i]) >= bfmap_thres) continue;
                }
                float thres = index_thres_for_matching + abs(img_index_left_sub_px[line_start_addr_offset+w] - w - img_index_right_sub_px[line_start_addr_offset+i] + i)/projector_area_ratio;
                if ((line_l[w]-thres <= line_r[i]) & (line_r[i] <= line_l[w])) {
                    if (most_corres_pts_l==-1) most_corres_pts_l = i;
                    else if (line_r[i] >= line_r[most_corres_pts_l]) most_corres_pts_l = i;
                    cnt_l += 1;
                    average_corres_position_in_thres_l += i;
                }
                else if ((line_l[w] <= line_r[i]) & (line_r[i] <= line_l[w]+thres)) {
                    if (most_corres_pts_r==-1) most_corres_pts_r = i;
                    else if (line_r[i] <= line_r[most_corres_pts_r]) most_corres_pts_r = i;
                    cnt_r += 1;
                    average_corres_position_in_thres_r += i;
                }
            }
        }
        // last_right_corres_point is invalid or not found most_corres_pts, expand the searching range and try searching again
        if (most_corres_pts_l == -1 & most_corres_pts_r == -1) {
            for (int i=0; i < width; i++) { 
                if (isnan(line_r[i])) continue;
                if (use_belief_map_checking_when_matching) {
                    float bfmap_thres = 10 + (belief_map_l[curr_pix_idx] + belief_map_r[line_start_addr_offset+i]) * 0.75;
                    if (abs(belief_map_l[curr_pix_idx] - belief_map_r[line_start_addr_offset+i]) >= bfmap_thres) continue;
                }
                // if (!belief_map_r[line_start_addr_offset+i]) continue;
                if ((line_l[w]-index_thres_for_matching <= line_r[i]) & (line_r[i] <= line_l[w])) {
                    if (most_corres_pts_l==-1) most_corres_pts_l = i;
                    else if (line_r[i] >= line_r[most_corres_pts_l]) most_corres_pts_l = i;
                    cnt_l += 1;
                    average_corres_position_in_thres_l += i;
                }
                else if ((line_l[w] <= line_r[i]) & (line_r[i] <= line_l[w]+index_thres_for_matching)) {
                    if (most_corres_pts_r==-1) most_corres_pts_r = i;
                    else if (line_r[i] <= line_r[most_corres_pts_r]) most_corres_pts_r = i;
                    cnt_r += 1;
                    average_corres_position_in_thres_r += i;
                }
            }
        }
        // get the right index
        float w_r = 0;
        bool outliers_flag = false;
        if (most_corres_pts_l == -1 & most_corres_pts_r == -1) continue;
        else if (most_corres_pts_l==-1) w_r = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_r]+0.2; // add 0.2 pix offset as we know it's on the right side
        else if (most_corres_pts_r==-1) w_r = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_l]-0.2;
        else {
            // get the interpo right index 'w_r'
            float left_pos = line_r[most_corres_pts_l], right_pos = line_r[most_corres_pts_r];
            float left_value = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_l], right_value = img_index_right_sub_px[line_start_addr_offset+most_corres_pts_r];
            if (right_pos-left_pos != 0) w_r = left_value + (right_value-left_value) * (line_l[w]-left_pos)/(right_pos-left_pos);
            else w_r = left_value;
        }
        if (cnt_l != 0) average_corres_position_in_thres_l = average_corres_position_in_thres_l / cnt_l;
        if (cnt_r != 0) average_corres_position_in_thres_r = average_corres_position_in_thres_r / cnt_r;
        // check possiblely outliers using max_allow_pixel_per_index and belief_map
        if (check_outliers==true & belief_map_r[line_start_addr_offset+(int)(w_r+0.499999)]==0) {  // & belief_map_r[line_start_addr_offset+(int)(w_r+0.499999)]==0
            if (most_corres_pts_l != -1 & abs((float)(most_corres_pts_l-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
            if (most_corres_pts_r != -1 & abs((float)(most_corres_pts_r-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
            if (average_corres_position_in_thres_l != 0 & abs((float)(average_corres_position_in_thres_l-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
            if (average_corres_position_in_thres_r != 0 & abs((float)(average_corres_position_in_thres_r-w_r)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
        }
        if (outliers_flag==true) continue;
        last_right_corres_point = (int)(w_r+0.499999);
        // get left index
        float w_l = img_index_left_sub_px[curr_pix_idx];
        // check possiblely left outliers
        if (check_outliers==true & belief_map_l[curr_pix_idx]==0) {
            for (int i=0; i < width; i++) {
                if ((line_l[w]-index_thres_for_outliers_checking <= line_l[i]) & (line_l[i] <= line_l[w]+index_thres_for_outliers_checking)) {
                    if (abs((float)(w_l-i)) > max_allow_pixel_per_index_for_outliers_checking) outliers_flag = true;
                }
            }
        }
        if (outliers_flag==true) continue;
        // get stereo diff and depth
        float stereo_diff = dmap_base[0] + w_l - w_r;
        if (stereo_diff > 0.000001) {
            float depth = fx[0] * baseline[0] / stereo_diff;
            if ((depth_cutoff_near < depth) & (depth < depth_cutoff_far)) depth_map[curr_pix_idx] = depth;
        }
    }
}

__global__ void optimize_dmap_using_sub_pixel_map(float *depth_map, float *optimized_depth_map, int *height_array, int *width_array, float *img_index_left_sub_px)
{
    // interpo for depth map using sub_pixel
    // this does not improve a lot on rendered data because no distortion and less stereo rectify for left camera, but useful for real captures
    int width = width_array[0];
    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int w = current_pix_idx % width;
    if (w == 0 | w == width-1) {
        optimized_depth_map[current_pix_idx] = depth_map[current_pix_idx];
        return;
    }

    float left_value = 0.0, right_value = 0.0;
    float left_pos = 0.0, right_pos = 0.0;
    float real_pos_for_current_depth = img_index_left_sub_px[current_pix_idx];
    if (depth_map[current_pix_idx] <= 0.00001) {
        if (depth_map[current_pix_idx-1] >= 0.00001 & depth_map[current_pix_idx+1] >= 0.00001) {
            right_pos = img_index_left_sub_px[current_pix_idx+1];
            right_value = depth_map[current_pix_idx+1];
            left_pos = img_index_left_sub_px[current_pix_idx-1];
            left_value = depth_map[current_pix_idx-1];
        }
    }
    else if (real_pos_for_current_depth >= w) {
        right_pos = real_pos_for_current_depth;
        right_value = depth_map[current_pix_idx];
        left_pos = img_index_left_sub_px[current_pix_idx-1];
        left_value = depth_map[current_pix_idx-1];
    }
    else {
        right_pos = img_index_left_sub_px[current_pix_idx+1];
        right_value = depth_map[current_pix_idx+1];
        left_pos = real_pos_for_current_depth;
        left_value = depth_map[current_pix_idx];
    }
    if (left_value >= 0.00001 & right_value >= 0.00001) {
        optimized_depth_map[current_pix_idx] = left_value + (right_value-left_value) * (w-left_pos)/(right_pos-left_pos);
    }
    else {
        optimized_depth_map[current_pix_idx] = 0.0;
    }
}

__global__ void flying_points_filter(float *depth_map, float *depth_map_raw, int *height_array, int *width_array, float *camera_kp, float *depth_filter_max_distance, int *depth_filter_minmum_points_in_checking_range, int *belief_map)
{
    // a point could be considered as not flying when: points in checking range below max_distance > minmum num 
    const bool use_3d_distance = false; // use 3D distance (slower but more precisely) or only distance of axis-z to check flying points
                                        // setting to false will save above 95% time compared with 3D distance checking, while can still remove most of the flying pts.
                                        // an example (3d vs only_z): render0000_2k avg error @ 10 mm thres: 0.1145mm vs 0.1152mm; cost time: 17ms vs 1ms; total time 80ms vs 66ms
    int height = height_array[0];
    int width = width_array[0];
    float max_distance = depth_filter_max_distance[0];
    int minmum_point_num_in_range = depth_filter_minmum_points_in_checking_range[0] + (width / 400) * (width / 400);
    float checking_range_in_meter = max_distance * 1.2;
    int checking_range_limit = width/50;
    float fx = camera_kp[0];
    float cx = camera_kp[2];
    float fy = camera_kp[1*3+1];
    float cy = camera_kp[1*3+2];

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map_raw[current_pix_idx];
    depth_map[current_pix_idx] = curr_pix_value;

    if (curr_pix_value != 0) {
        float point_x = curr_pix_value * (w - cx) / fx;
        float point_y = curr_pix_value * (h - cy) / fy;
        int checking_range_in_pix_x = (int)(checking_range_in_meter * fx / curr_pix_value);
        int checking_range_in_pix_y = (int)(checking_range_in_meter * fy / curr_pix_value);
        checking_range_in_pix_x = min(checking_range_in_pix_x, checking_range_limit);
        checking_range_in_pix_y = min(checking_range_in_pix_y, checking_range_limit);
        int is_not_flying_point_flag = 0;
        
        for (unsigned int i = max(0, h-checking_range_in_pix_y); i < min(height, h+checking_range_in_pix_y+1); i++) {
            int line_i_offset = i * width;
            for (unsigned int j = max(0, w-checking_range_in_pix_x); j < min(width, w+checking_range_in_pix_x+1); j++) {
                float checking_pix_value = depth_map_raw[line_i_offset + j];
                float z_diff = abs(curr_pix_value - checking_pix_value);
                if (checking_pix_value != 0.0 & z_diff < max_distance) {
                    if (use_3d_distance) {
                        float curr_x = checking_pix_value * (j - cx) / fx;
                        float curr_y = checking_pix_value * (i - cy) / fy;
                        float x_diff = curr_x - point_x, y_diff = curr_y - point_y;
                        float distance = (x_diff)*(x_diff) + (y_diff)*(y_diff) + (z_diff)*(z_diff);
                        if (distance < max_distance*max_distance) is_not_flying_point_flag += 1;
                    }
                    else is_not_flying_point_flag += 1; 
                }
            }
            if (is_not_flying_point_flag > minmum_point_num_in_range) break;
        }
        if (is_not_flying_point_flag <= minmum_point_num_in_range) depth_map[current_pix_idx] = 0.0;
    }
}

__global__ void depth_filter(float *depth_map, int *height_array, int *width_array, int *depth_filter_max_length, float *depth_filter_unvalid_thres, int *belief_map)
{
    // a point could be considered as not flying when: points in checking range below max_distance > minmum num 
    int height = height_array[0];
    int width = width_array[0];
    int filter_max_length = depth_filter_max_length[0];
    float filter_thres = depth_filter_unvalid_thres[0];
    const float filter_weights[6] = {1.0, 0.8, 0.6, 0.5, 0.4, 0.2};

    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    int h = blockIdx.x / 4;
    int w = current_pix_idx % width;
    float curr_pix_value = depth_map[current_pix_idx];
    if (curr_pix_value != 0) {
        int line_start_addr_offset = h * width;
        // horizontal
        float left_weight = 0.0, right_weight = 0.0, depth_sum = curr_pix_value*filter_weights[0];
        bool stop_flag = false;
        for (int i=1; i< filter_max_length+1; i++) {
            int l_idx = w-i, r_idx = w+i;
            // if (belief_map[current_pix_idx] == 1) filter_thres = depth_filter_unvalid_thres[0];
            // else filter_thres = depth_filter_unvalid_thres[0] * 3;
            if(depth_map[line_start_addr_offset+l_idx] != 0 & depth_map[line_start_addr_offset+r_idx] != 0 & l_idx > 0 & r_idx < width & \
                abs(depth_map[line_start_addr_offset+l_idx] - curr_pix_value) < filter_thres & abs(depth_map[line_start_addr_offset+r_idx] - curr_pix_value) < filter_thres) {
                left_weight += filter_weights[i];
                right_weight += filter_weights[i];
                depth_sum += (depth_map[line_start_addr_offset+r_idx] + depth_map[line_start_addr_offset+l_idx]) * filter_weights[i];
            }
            else {
                stop_flag = true; 
                break;
            }
        }
        if (!stop_flag) depth_map[current_pix_idx] = depth_sum / (filter_weights[0] + left_weight + right_weight);
        __threadfence();  // make sure the modification to depth_map is visiable for all other threads
        __syncthreads();
        // vertical
        curr_pix_value = depth_map[current_pix_idx];
        left_weight = 0.0, right_weight = 0.0, depth_sum = curr_pix_value*filter_weights[0];
        stop_flag = false;
        for (int i=1; i< filter_max_length+1; i++) {
            int l_idx = h-i, r_idx = h+i;
            //if (belief_map[current_pix_idx] == 1) filter_thres = depth_filter_unvalid_thres[0];
            //else filter_thres = depth_filter_unvalid_thres[0] * 3;
            if(depth_map[l_idx*width+w] != 0 & depth_map[r_idx*width+w] != 0 & l_idx > 0 & r_idx < height & \
                abs(depth_map[l_idx*width+w] - curr_pix_value) < filter_thres & abs(depth_map[r_idx*width+w] - curr_pix_value) < filter_thres) {
                left_weight += filter_weights[i];
                right_weight += filter_weights[i];
                depth_sum += (depth_map[r_idx*width+w] + depth_map[l_idx*width+w]) * filter_weights[i];
            }
            else {
                stop_flag = true; 
                break;
            }
        }
        if (!stop_flag) depth_map[current_pix_idx] = depth_sum / (filter_weights[0] + left_weight + right_weight);
    }
}

__global__ void convert_dmap_to_mili_meter(float *depth_map)
{
    int current_pix_idx = threadIdx.x + blockIdx.x*blockDim.x;
    depth_map[current_pix_idx] = 1000.0*depth_map[current_pix_idx];
}
